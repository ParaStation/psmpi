#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 */

#include <mpi.h>
#include <stdio.h>
#include <assert.h>

const int N = 1000000;
const int a = 2.0;
const float x_val = 1.0f;
const float y_val = 2.0f;
const float exp_result = 4.0f;

static int check_result(float *y)
{
    float maxError = 0.0f;
    int errs = 0;
    for (int i = 0; i < N; i++) {
        if (abs(y[i] - exp_result) > 0.01) {
            errs++;
            maxError = max(maxError, abs(y[i] - exp_result));
        }
    }
    if (errs > 0) {
        printf("%d errors, Max error: %f\n", errs, maxError);
    }
    return errs;
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

static int need_progress_thread = 0;
static void parse_args(int argc, char **argv)
{
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-progress-thread") == 0) {
            need_progress_thread = 1;
        }
    }
}

int main(int argc, char **argv)
{
    int errs = 0;

    parse_args(argc, argv);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int mpi_errno;
    int rank, size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (size < 2) {
        printf("This test require 2 processes\n");
        exit(1);
    }

    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    MPI_Info info;
    MPI_Info_create(&info);
    MPI_Info_set(info, "type", "hipStream_t");
    MPIX_Info_set_hex(info, "value", &stream, sizeof(stream));

    MPIX_Stream mpi_stream;
    MPIX_Stream_create(info, &mpi_stream);

    MPI_Info_free(&info);

    if (need_progress_thread) {
        MPIX_Start_progress_thread(mpi_stream);
    }

    MPI_Comm stream_comm;
    MPIX_Stream_comm_create(MPI_COMM_WORLD, mpi_stream, &stream_comm);

    /* Rank 0 sends x data to Rank 1, Rank 1 performs a * x + y and checks result */
    if (rank == 0) {
        for (int i = 0; i < N; i++) {
            x[i] = x_val;
        }
        hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Send_enqueue(d_x, N, MPI_FLOAT, 1, 0, stream_comm);
        assert(mpi_errno == MPI_SUCCESS);
    } else if (rank == 1) {
        for (int i = 0; i < N; i++) {
            y[i] = y_val;
        }
        hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Recv_enqueue(d_x, N, MPI_FLOAT, 0, 0, stream_comm, MPI_STATUS_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);

        saxpy<<<(N+255)/256, 256, 0, stream>>>(N, a, d_x, d_y);

        hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
        errs += check_result(y);
    }

    /* Test again with MPIX_Isend_enqueue and MPIX_Wait_enqueue */
    if (rank == 0) {
        for (int i = 0; i < N; i++) {
            x[i] = x_val;
        }
        /* we are directly sending from x in this test */
        MPI_Request req;
        mpi_errno = MPIX_Isend_enqueue(x, N, MPI_FLOAT, 1, 0, stream_comm, &req);
        assert(mpi_errno == MPI_SUCCESS);
        /* req won't reset to MPI_REQUEST_NULL, but user shouldn't use it afterward */
        mpi_errno = MPIX_Wait_enqueue(&req, MPI_STATUS_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);
    } else if (rank == 1) {
        /* reset d_x, d_y */
        for (int i = 0; i < N; i++) {
            x[i] = 0.0;
            y[i] = y_val;
        }
        hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream);

        MPI_Request req;
        mpi_errno = MPIX_Irecv_enqueue(d_x, N, MPI_FLOAT, 0, 0, stream_comm, &req);
        assert(mpi_errno == MPI_SUCCESS);
        mpi_errno = MPIX_Waitall_enqueue(1, &req, MPI_STATUSES_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);

        saxpy<<<(N+255)/256, 256, 0, stream>>>(N, a, d_x, d_y);

        hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
        errs += check_result(y);
    }

    if (need_progress_thread) {
        MPIX_Stop_progress_thread(mpi_stream);
    }

    MPI_Comm_free(&stream_comm);
    MPIX_Stream_free(&mpi_stream);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    hipStreamDestroy(stream);
    MPI_Finalize();

    if (rank == 1 && errs == 0) {
        printf("No Errors\n");
    }
    return errs;
}

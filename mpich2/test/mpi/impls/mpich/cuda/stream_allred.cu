/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 */

#include <mpi.h>
#include <stdio.h>
#include <assert.h>

int rank, size;

#define CHECK_RESULT(i, result, expected, msg) \
    do { \
        if (result != expected) { \
            printf("%s: i = %d, expect %d, got %d\n", msg, i, expected, result); \
            errs++; \
        } \
    } while (0)

static int test_1(MPI_Comm comm, hipStream_t stream, bool do_inplace)
{
    int errs = 0;
    int mpi_errno;
#define N 10    
    /* TEST 1 - MPI_INT */
    int buf[N];
    void *d_buf, *d_result_buf;
    hipMalloc(&d_buf, sizeof(buf));
    hipMalloc(&d_result_buf, sizeof(buf));

    for (int i = 0; i < N; i++) {
        buf[i] = rank;
    }

    int expected_sum = size * (size - 1) / 2;

    const void *sendbuf;
    void *recvbuf;
    if (do_inplace) {
        hipMemcpyAsync(d_result_buf, buf, sizeof(buf), hipMemcpyHostToDevice, stream);
        sendbuf = MPI_IN_PLACE;
        recvbuf = d_result_buf;
    } else {
        hipMemcpyAsync(d_buf, buf, sizeof(buf), hipMemcpyHostToDevice, stream);
        sendbuf = d_buf;
        recvbuf = d_result_buf;
    }
    mpi_errno = MPIX_Allreduce_enqueue(sendbuf, recvbuf, N, MPI_INT, MPI_SUM, comm);
    assert(mpi_errno == MPI_SUCCESS);
    hipMemcpyAsync(buf, d_result_buf, sizeof(buf), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipFree(d_buf);
    hipFree(d_result_buf);

    const char *test_name = "Test 1";
    if (do_inplace) {
        test_name = "TEST 1 (MPI_IN_PLACE)";
    }
    for (int i = 0; i < N; i++) {
        CHECK_RESULT(i, buf[i], expected_sum, test_name);
    }
#undef N
    return errs;
}

static int test_2(MPI_Comm comm, hipStream_t stream, bool do_inplace)
{
    int errs = 0;
    int mpi_errno;
#define N 10    
    /* TEST 2 - Pairtype (non-contig) */
    struct {
        short a;
        int b;
    } buf[N];
    void *d_buf, *d_result_buf;
    hipMalloc(&d_buf, sizeof(buf));
    hipMalloc(&d_result_buf, sizeof(buf));

    for(int i = 0; i < N; i++) {
        /* MINLOC result should be {0, i % size} */
        if (i % size == rank) {
            buf[i].a = 0;
        } else {
            buf[i].a = rank + 1;
        }
        buf[i].b = rank;
    }

    const void *sendbuf;
    void *recvbuf;
    if (do_inplace) {
        hipMemcpyAsync(d_result_buf, buf, sizeof(buf), hipMemcpyHostToDevice, stream);
        sendbuf = MPI_IN_PLACE;
        recvbuf = d_result_buf;
    } else {
        hipMemcpyAsync(d_buf, buf, sizeof(buf), hipMemcpyHostToDevice, stream);
        sendbuf = d_buf;
        recvbuf = d_result_buf;
    }
    mpi_errno = MPIX_Allreduce_enqueue(sendbuf, recvbuf, N, MPI_SHORT_INT, MPI_MINLOC, comm);
    assert(mpi_errno == MPI_SUCCESS);
    hipMemcpyAsync(buf, d_result_buf, sizeof(buf), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    const char *test_name = "Test 2";
    if (do_inplace) {
        test_name = "TEST 2 (MPI_IN_PLACE)";
    }
    for (int i = 0; i < N; i++) {
        CHECK_RESULT(i, buf[i].a, 0, test_name);
        CHECK_RESULT(i, buf[i].b, i % size, test_name);
    }
#undef N
    return errs;
}

int main(void)
{
    int errs = 0;

    hipStream_t stream;
    hipStreamCreate(&stream);

    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    MPI_Info info;
    MPI_Info_create(&info);
    MPI_Info_set(info, "type", "hipStream_t");
    MPIX_Info_set_hex(info, "value", &stream, sizeof(stream));

    MPIX_Stream mpi_stream;
    MPIX_Stream_create(info, &mpi_stream);

    MPI_Info_free(&info);

    MPI_Comm stream_comm;
    MPIX_Stream_comm_create(MPI_COMM_WORLD, mpi_stream, &stream_comm);

    errs += test_1(stream_comm, stream, false);
    errs += test_1(stream_comm, stream, true);  /* MPI_IN_PLACE */
    errs += test_2(stream_comm, stream, false);
    errs += test_2(stream_comm, stream, true);  /* MPI_IN_PLACE */

    /* clean up */
    MPI_Comm_free(&stream_comm);
    MPIX_Stream_free(&mpi_stream);

    hipStreamDestroy(stream);

    int tot_errs;
    MPI_Reduce(&errs, &tot_errs, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);
    if (rank == 0) {
        if (tot_errs == 0) {
            printf("No Errors\n");
        } else {
            printf("%d Errors\n", tot_errs);
        }
    }

    MPI_Finalize();
    return errs;
}

#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 */

#include "mpi.h"
#include <stdio.h>
#include <math.h>

#define THREADS_PER_BLOCK 256

__device__ double f(double a)
{
    return (4.0 / (1.0 + a * a));
}

__global__ void do_sum(int n, double h, int stride, double *sum) {
    int idx = 1 + (blockDim.x * blockIdx.x + threadIdx.x) + stride;
    __shared__ double block_sum;

    if (threadIdx.x == 0) {
        block_sum = 0.0;
    }
    __syncthreads();

    /* compute rectangles and add to block sum */
    if (idx <= n) {
        double x = h * ((double) idx - 0.5);
        atomicAdd(&block_sum, f(x));
    }

    /* add block sum to total */
    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(sum, block_sum * h);
    }
}

int main(int argc, char *argv[])
{
    int n, myid, numprocs;
    double PI25DT = 3.141592653589793238462643;
    double pi, h;
    double *sum;
    double startwtime = 0.0, endwtime;
    int namelen;
    char processor_name[MPI_MAX_PROCESSOR_NAME];

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);
    MPI_Get_processor_name(processor_name, &namelen);

    fprintf(stdout, "Process %d of %d is on %s\n", myid, numprocs, processor_name);
    fflush(stdout);

    hipMalloc((void **)&sum, sizeof(double));

    n = 10000;
    if (myid == 0)
        startwtime = MPI_Wtime();

    MPI_Bcast(&n, 1, MPI_INT, 0, MPI_COMM_WORLD);

    h = 1.0 / (double) n;
    int blocks = (n + (THREADS_PER_BLOCK * numprocs - 1)) / (THREADS_PER_BLOCK * numprocs);
    int stride = blocks * THREADS_PER_BLOCK * myid;

    /* compute partial sum using the GPU */
    do_sum<<<blocks, THREADS_PER_BLOCK>>>(n, h, stride, sum);
    hipDeviceSynchronize();

    MPI_Reduce(sum, &pi, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);

    if (myid == 0) {
        endwtime = MPI_Wtime();
        printf("pi is approximately %.16f, Error is %.16f\n", pi, fabs(pi - PI25DT));
        printf("wall clock time = %f\n", endwtime - startwtime);
        fflush(stdout);
    }

    hipFree(sum);

    MPI_Finalize();
    return 0;
}

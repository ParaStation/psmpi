#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent));
}

void yaksuri_cudai_pack_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    *((double *) (void *) (dbuf + x0 * extent)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hvector_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2));
}

void yaksuri_cudai_pack_hvector_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_hvector_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_blkhindx_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2));
}

void yaksuri_cudai_pack_blkhindx_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_blkhindx_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_blkhindx_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    *((double *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_blkhindx_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_blkhindx_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hindexed_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2));
}

void yaksuri_cudai_pack_hindexed_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hindexed_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hindexed_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    *((double *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_hindexed_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hindexed_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_contig_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1));
}

void yaksuri_cudai_pack_contig_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_contig_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_contig_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    *((double *) (void *) (dbuf + x0 * extent + x1 * stride1)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_contig_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_contig_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_resized_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    *((double *) (void *) (dbuf + idx * sizeof(double))) = *((const double *) (const void *) (sbuf + x0 * extent));
}

void yaksuri_cudai_pack_resized_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_resized_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_resized_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    *((double *) (void *) (dbuf + x0 * extent)) = *((const double *) (const void *) (sbuf + idx * sizeof(double)));
}

void yaksuri_cudai_unpack_resized_resized_resized_double(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_resized_resized_resized_double,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}


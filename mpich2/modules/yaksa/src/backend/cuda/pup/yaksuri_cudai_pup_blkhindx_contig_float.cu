#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.contig.count;
    
    uintptr_t x3 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t stride2 = md->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2));
}

void yaksuri_cudai_pack_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.contig.count;
    
    uintptr_t x3 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t stride2 = md->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hvector_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    intptr_t stride3 = md->u.hvector.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3));
}

void yaksuri_cudai_pack_hvector_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    intptr_t stride3 = md->u.hvector.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_hvector_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_blkhindx_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t *array_of_displs2 = md->u.blkhindx.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    intptr_t stride3 = md->u.blkhindx.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3));
}

void yaksuri_cudai_pack_blkhindx_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_blkhindx_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_blkhindx_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t *array_of_displs2 = md->u.blkhindx.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    intptr_t stride3 = md->u.blkhindx.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_blkhindx_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_blkhindx_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hindexed_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t *array_of_displs2 = md->u.hindexed.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    intptr_t stride3 = md->u.hindexed.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3));
}

void yaksuri_cudai_pack_hindexed_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hindexed_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hindexed_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.blocklength;
    uintptr_t x4 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x5 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t *array_of_displs2 = md->u.hindexed.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    intptr_t stride3 = md->u.hindexed.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3 + x5 * stride3)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_hindexed_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hindexed_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_contig_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.count;
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.blocklength;
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.blkhindx.array_of_displs;
    intptr_t stride3 = md->u.contig.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.contig.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * extent3 + x4 * stride3));
}

void yaksuri_cudai_pack_contig_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_contig_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_contig_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.count;
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.blocklength;
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.blkhindx.array_of_displs;
    intptr_t stride3 = md->u.contig.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.contig.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * extent3 + x4 * stride3)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_contig_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_contig_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_resized_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x3 = res;
    
    intptr_t *array_of_displs2 = md->u.resized.child->u.blkhindx.array_of_displs;
    intptr_t stride3 = md->u.resized.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.resized.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + array_of_displs2[x1] + x2 * extent3 + x3 * stride3));
}

void yaksuri_cudai_pack_resized_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_resized_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_resized_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.blkhindx.child->u.contig.count;
    
    uintptr_t x3 = res;
    
    intptr_t *array_of_displs2 = md->u.resized.child->u.blkhindx.array_of_displs;
    intptr_t stride3 = md->u.resized.child->u.blkhindx.child->u.contig.child->extent;
    uintptr_t extent3 = md->u.resized.child->u.blkhindx.child->extent;
    *((float *) (void *) (dbuf + x0 * extent + array_of_displs2[x1] + x2 * extent3 + x3 * stride3)) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_resized_blkhindx_contig_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_resized_blkhindx_contig_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}


#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_hvector_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    *((float *) (void *) (dbuf + idx * sizeof(float))) = *((const float *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * sizeof(float)));
}

void yaksuri_cudai_pack_hvector_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_float(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    *((float *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * sizeof(float))) = *((const float *) (const void *) (sbuf + idx * sizeof(float)));
}

void yaksuri_cudai_unpack_hvector_float(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_float,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}


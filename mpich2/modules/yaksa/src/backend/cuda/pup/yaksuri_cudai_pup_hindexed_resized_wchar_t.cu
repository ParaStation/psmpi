#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2));
}

void yaksuri_cudai_pack_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hvector_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.hvector.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hvector.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hvector.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.hvector.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * extent3));
}

void yaksuri_cudai_pack_hvector_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.hvector.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hvector.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hvector.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.hvector.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * extent3)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_hvector_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_blkhindx_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.blkhindx.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.blkhindx.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.blkhindx.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.blkhindx.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t *array_of_displs2 = md->u.blkhindx.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3));
}

void yaksuri_cudai_pack_blkhindx_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_blkhindx_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_blkhindx_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.blkhindx.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.blkhindx.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.blkhindx.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.blkhindx.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t *array_of_displs2 = md->u.blkhindx.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_blkhindx_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_blkhindx_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hindexed_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.hindexed.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t *array_of_displs2 = md->u.hindexed.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3));
}

void yaksuri_cudai_pack_hindexed_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hindexed_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hindexed_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hindexed.count;
    
    uintptr_t x3;
    for (int i = 0; i < md->u.hindexed.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x3 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t *array_of_displs2 = md->u.hindexed.child->u.hindexed.array_of_displs;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + array_of_displs2[x3] + x4 * extent3)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_hindexed_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hindexed_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_contig_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (int i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    uintptr_t extent3 = md->u.contig.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * extent3));
}

void yaksuri_cudai_pack_contig_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_contig_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_contig_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (int i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    uintptr_t extent3 = md->u.contig.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * extent3)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_contig_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_contig_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_resized_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.resized.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.resized.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.resized.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.resized.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs2 = md->u.resized.child->u.hindexed.array_of_displs;
    uintptr_t extent3 = md->u.resized.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))) = *((const wchar_t *) (const void *) (sbuf + x0 * extent + array_of_displs2[x1] + x2 * extent3));
}

void yaksuri_cudai_pack_resized_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_resized_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_resized_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.resized.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.resized.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.resized.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.resized.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res;
    
    intptr_t *array_of_displs2 = md->u.resized.child->u.hindexed.array_of_displs;
    uintptr_t extent3 = md->u.resized.child->u.hindexed.child->extent;
    *((wchar_t *) (void *) (dbuf + x0 * extent + array_of_displs2[x1] + x2 * extent3)) = *((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t)));
}

void yaksuri_cudai_unpack_resized_hindexed_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_resized_hindexed_resized_wchar_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}


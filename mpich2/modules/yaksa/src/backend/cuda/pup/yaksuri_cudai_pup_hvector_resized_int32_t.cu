#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2));
}

void yaksuri_cudai_pack_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hvector_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t stride2 = md->u.hvector.child->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + x3 * stride2 + x4 * extent3));
}

void yaksuri_cudai_pack_hvector_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hvector_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hvector_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t stride2 = md->u.hvector.child->u.hvector.stride;
    uintptr_t extent2 = md->u.hvector.child->extent;
    uintptr_t extent3 = md->u.hvector.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + x3 * stride2 + x4 * extent3)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_hvector_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hvector_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_blkhindx_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t stride2 = md->u.blkhindx.child->u.hvector.stride;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2 + x4 * extent3));
}

void yaksuri_cudai_pack_blkhindx_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_blkhindx_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_blkhindx_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.blkhindx.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.blkhindx.array_of_displs;
    intptr_t stride2 = md->u.blkhindx.child->u.hvector.stride;
    uintptr_t extent2 = md->u.blkhindx.child->extent;
    uintptr_t extent3 = md->u.blkhindx.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2 + x4 * extent3)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_blkhindx_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_blkhindx_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_hindexed_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t stride2 = md->u.hindexed.child->u.hvector.stride;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2 + x4 * extent3));
}

void yaksuri_cudai_pack_hindexed_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_hindexed_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_hindexed_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.count;
    
    uintptr_t x1;
    for (int i = 0; i < md->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.hindexed.array_of_blocklengths[i] *
                                 md->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x1 = i;
                    res %= in_elems;
                    inner_elements = md->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hvector.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hindexed.child->u.hvector.blocklength;
    uintptr_t x4 = res;
    
    intptr_t *array_of_displs1 = md->u.hindexed.array_of_displs;
    intptr_t stride2 = md->u.hindexed.child->u.hvector.stride;
    uintptr_t extent2 = md->u.hindexed.child->extent;
    uintptr_t extent3 = md->u.hindexed.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + array_of_displs1[x1] + x2 * extent2 + x3 * stride2 + x4 * extent3)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_hindexed_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_hindexed_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_contig_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hvector.count;
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hvector.blocklength;
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t stride2 = md->u.contig.child->u.hvector.stride;
    uintptr_t extent3 = md->u.contig.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * stride2 + x3 * extent3));
}

void yaksuri_cudai_pack_contig_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_contig_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_contig_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hvector.count;
    
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hvector.blocklength;
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t stride2 = md->u.contig.child->u.hvector.stride;
    uintptr_t extent3 = md->u.contig.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * stride2 + x3 * extent3)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_contig_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_contig_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_pack_resized_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride2 = md->u.resized.child->u.hvector.stride;
    uintptr_t extent3 = md->u.resized.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))) = *((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride2 + x2 * extent3));
}

void yaksuri_cudai_pack_resized_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_resized_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

__global__ void yaksuri_cudai_kernel_unpack_resized_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.resized.child->u.hvector.blocklength;
    uintptr_t x2 = res;
    
    intptr_t stride2 = md->u.resized.child->u.hvector.stride;
    uintptr_t extent3 = md->u.resized.child->u.hvector.child->extent;
    *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride2 + x2 * extent3)) = *((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t)));
}

void yaksuri_cudai_unpack_resized_hvector_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, int device)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_resized_hvector_resized_int32_t,
        dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, yaksuri_cudai_global.stream[device]);
    YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
}

